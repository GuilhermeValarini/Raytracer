// [header]
// A very basic raytracer example.
// [/header]
// [compile]
// c++ -o raytracer -O3 -Wall raytracer.cpp
// [/compile]
// [ignore]
// Copyright (C) 2012  www.scratchapixel.com
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
// [/ignore]

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <vector>
#include <iostream>
#include <cassert>
#include <cstring>
#include <sys/time.h>

#if defined __linux__ || defined __APPLE__
// "Compiled for Linux
#else
// Windows doesn't define these values by default, Linux does
#define M_PI 3.141592653589793
#define INFINITY 1e8
#endif

#ifndef TILE_WIDTH
#define TILE_WIDTH 8
#endif

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

class UnifiedMemoryClass
{
public:
    void* operator new(size_t len) {
        void *ptr;
        hipMallocManaged(&ptr, len);
        return ptr;
    }

    void operator delete(void *ptr) {
        hipFree(ptr);
    }

    void* operator new[](size_t len) {
        void *ptr;
        hipMallocManaged(&ptr, len);
        return ptr;
    }
};

class Vec3 : public UnifiedMemoryClass
{
public:
    float x, y, z;
    __device__ __host__ Vec3() : x(float(0)), y(float(0)), z(float(0)) {}
    __device__ __host__ Vec3(float xx) : x(xx), y(xx), z(xx) {}
    __device__ __host__ Vec3(float xx, float yy, float zz) : x(xx), y(yy), z(zz) {}
    __device__ Vec3& normalize()
    {
        float nor2 = length2();
        if (nor2 > 0) {
            float invNor = 1 / sqrt(nor2);
            x *= invNor, y *= invNor, z *= invNor;
        }
        return *this;
    }
    __device__ Vec3 operator * (const float &f) const { return Vec3(x * f, y * f, z * f); }
    __device__ static Vec3 mult (const float &f, const Vec3 v) {
        return Vec3(v.x * f, v.y * f, v.z * f);
    }
    __device__ Vec3 operator * (const Vec3 &v) const { return Vec3(x * v.x, y * v.y, z * v.z); }
    __device__ static Vec3 mult (const Vec3 &v1, const Vec3 &v2) {
        return Vec3(v1.x * v2.x, v1.y * v2.y, v1.z * v2.z);
    }
    __device__ float dot(const Vec3 &v) const { return x * v.x + y * v.y + z * v.z; }
    __device__ static float dot(const Vec3 &v1, const Vec3 &v2) {
        return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
    }
    __device__ Vec3 operator - (const Vec3 &v) const { return Vec3(x - v.x, y - v.y, z - v.z); }
    __device__ static Vec3 sub (const Vec3 &v1, const Vec3 &v2) {
        return Vec3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
    }
    __device__ Vec3 operator + (const Vec3 &v) const { return Vec3(x + v.x, y + v.y, z + v.z); }
    __device__ static Vec3 add (const Vec3 &v1, const Vec3 &v2) {
        return Vec3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
    }
    __device__ Vec3& operator += (const Vec3 &v) { x += v.x, y += v.y, z += v.z; return *this; }
    __device__ Vec3& operator *= (const Vec3 &v) { x *= v.x, y *= v.y, z *= v.z; return *this; }
    __device__ Vec3 operator - () const { return Vec3(-x, -y, -z); }
    __device__ static Vec3 neg (const Vec3 &v) {
        return Vec3(-v.x, -v.y, -v.z);
    }
    __device__ float length2() const { return x * x + y * y + z * z; }
    __device__ static float length2(const Vec3 &v) {
        return v.x * v.x + v.y * v.y + v.z * v.z;
    }
    __device__ float length() const { return sqrt(length2()); }
    __device__ static float length(const Vec3 &v) {
        return sqrt(length2(v));
    }
    __host__ friend std::ostream & operator << (std::ostream &os, const Vec3 &v)
    {
        os << "[" << v.x << " " << v.y << " " << v.z << "]";
        return os;
    }
};

__device__ Vec3 sub (const Vec3 &v1, const Vec3 &v2) {
    return Vec3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

typedef Vec3 Vec3f;
typedef Vec3 RGB;

class Sphere : public UnifiedMemoryClass
{
public:
    Vec3f center;                           /// position of the sphere
    float radius, radius2;                  /// sphere radius and radius^2
    Vec3f surfaceColor, emissionColor;      /// surface color and emission (light)
    float reflection;         /// surface transparency and reflectivity
    __device__ __host__ Sphere(
        const Vec3f &c,
        const float &r,
        const Vec3f &sc,
        const float &refl = 0,
        const Vec3f &ec = 0) :
        center(c), radius(r), radius2(r * r), surfaceColor(sc), emissionColor(ec),
        reflection(refl)
    { /* empty */ }
    __device__ __host__ Sphere(){}
    //[comment]
    // Compute a ray-sphere intersection using the geometric solution
    //[/comment]
    __device__ bool intersect(const Vec3f &rayorig, const Vec3f &raydir, float &t0, float &t1) const
    {
        Vec3f l = center - rayorig;
        float tca = l.dot(raydir);
        if (tca < 0) return false;
        float d2 = l.dot(l) - tca * tca;
        if (d2 > radius2) return false;
        float thc = sqrtf(radius2 - d2);
        t0 = tca - thc;
        t1 = tca + thc;

        return true;
    }
};

//[comment]
// This variable controls the maximum recursion depth
//[/comment]
#ifndef MAX_RAY_DEPTH
#define MAX_RAY_DEPTH 10
#endif

__device__ float mix(const float &a, const float &b, const float &mix)
{
    return b * mix + a * (1 - mix);
}

//[comment]
// This is the main trace function. It takes a ray as argument (defined by its origin
// and direction). We test if this ray intersects any of the geometry in the scene.
// If the ray intersects an object, we compute the intersection point, the normal
// at the intersection point, and shade this point using this information.
// Shading depends on the surface property (is it transparent, reflective, diffuse).
// The function returns a color for the ray. If the ray intersects an object that
// is the color of the object at the intersection point, otherwise it returns
// the background color.
//[/comment]
__global__ void trace(
    const Sphere* spheres,
    const unsigned n,
    Vec3f *image,
    unsigned width,
    unsigned height)
{
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned tid = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned blockSize = blockDim.x * blockDim.y;

    extern __shared__ Sphere s_spheres[];

    if (tid < n) {
        s_spheres[tid] = spheres[tid];
    }

    if (blockSize < n && tid == 0) {
        for (unsigned i = blockSize; i < n; i++) {
            s_spheres[i] = spheres[i];
        }
    }

    __syncthreads();

    if (x < width && y < height) {
        float fov = 30;
        float aspectratio = width / float(height);
        float angle = tan(M_PI * 0.5 * fov / 180.);
        float xx = (2 * ((x + 0.5) / float(width)) - 1) * angle * aspectratio;
        float yy = (1 - 2 * ((y + 0.5) / float(height))) * angle;
        Vec3f rayorig(0);
        Vec3f raydir(xx, yy, -1);
        raydir.normalize();

        int depth = 0;
        Vec3f reflectionMultStack[MAX_RAY_DEPTH+1];
        Vec3f reflectionAddStack[MAX_RAY_DEPTH+1];
        Vec3f result;

        while (true) {
            //if (raydir.length() != 1) std::cerr << "Error " << raydir << std::endl;
            float tnear = INFINITY;
            const Sphere* sphere = NULL;
            // find intersection of this ray with the sphere in the scene
            for (unsigned i = 0; i < n; ++i) {
                float t0 = INFINITY, t1 = INFINITY;
                if (s_spheres[i].intersect(rayorig, raydir, t0, t1)) {
                    if (t0 < 0) t0 = t1;
                    if (t0 < tnear) {
                        tnear = t0;
                        sphere = &s_spheres[i];
                    }
                }
            }
            // if there's no intersection return black or background color
            if (!sphere) {
                //return Vec3f(2);
                result = Vec3f(2);
                break;
            }
            Vec3f surfaceColor = 0; // color of the ray/surfaceof the object intersected by the ray
            Vec3f phit = rayorig + raydir * tnear; // point of intersection
            Vec3f nhit = phit - sphere->center; // normal at the intersection point
            nhit.normalize(); // normalize normal direction
            // If the normal and the view direction are not opposite to each other
            // reverse the normal direction. That also means we are inside the sphere so set
            // the inside bool to true. Finally reverse the sign of IdotN which we want
            // positive.
            float bias = 1e-4; // add some bias to the point from which we will be tracing
            if (raydir.dot(nhit) > 0) nhit = -nhit;
            if (sphere->reflection > 0 && depth < MAX_RAY_DEPTH) {
                float facingratio = -raydir.dot(nhit);
                // change the mix value to tweak the effect
                float fresneleffect = mix(pow(1 - facingratio, 3), 1, 0.1);
                // compute reflection direction (not need to normalize because all vectors
                // are already normalized)
                Vec3f refldir = raydir - nhit * 2 * raydir.dot(nhit);
                refldir.normalize();
                reflectionMultStack[depth] = sphere->surfaceColor * fresneleffect;
                reflectionAddStack[depth] = sphere->emissionColor;
                rayorig = phit + nhit * bias;
                raydir = refldir;
                depth++;
                continue;
            }
            else {
                // it's a diffuse object, no need to raytrace any further
                for (unsigned i = 0; i < n; ++i) {
                    if (s_spheres[i].emissionColor.x > 0) {
                        // this is a light
                        Vec3f transmission = 1;
                        Vec3f lightDirection = s_spheres[i].center - phit;
                        lightDirection.normalize();
                        for (unsigned j = 0; j < n; ++j) {
                            if (i != j) {
                                float t0, t1;
                                if (s_spheres[j].intersect(phit + nhit * bias, lightDirection, t0, t1)) {
                                    transmission = 0;
                                    break;
                                }
                            }
                        }
                        surfaceColor += sphere->surfaceColor * transmission *
                        fmaxf(float(0), nhit.dot(lightDirection)) * s_spheres[i].emissionColor;
                        // return surfaceColor + sphere->emissionColor;
                    }
                }
                result = surfaceColor + sphere->emissionColor;
                break;
            }
        }

        for (depth = depth - 1; depth >= 0; depth--) {
            result = result * reflectionMultStack[depth] + reflectionAddStack[depth];
        }

        image[y*width+x] = result;
    }
}

//[comment]
// Main rendering function. We compute a camera ray for each pixel of the image
// trace it and return a color. If the ray hits a sphere, we return the color of the
// sphere at the intersection point, else we return the background color.
//[/comment]
void render(Vec3f* image, unsigned width, unsigned height, const Sphere* spheres, const unsigned n)
{
    dim3 dimGrid(ceil((float) width / TILE_WIDTH), ceil((float)height / TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

    trace<<<dimGrid, dimBlock, n * sizeof(Sphere)>>>(spheres, n, image, width, height);
    hipDeviceSynchronize();
}

void save(const char* filename, Vec3f* image, unsigned width, unsigned height) {
    // Save result to a PPM image (keep these flags if you compile under Windows)
    std::ofstream ofs(filename, std::ios::out | std::ios::binary);
    ofs << "P6\n" << width << " " << height << "\n255\n";
    for (unsigned i = 0; i < width * height; ++i) {
        ofs << (unsigned char)(std::min(float(1), image[i].x) * 255) <<
               (unsigned char)(std::min(float(1), image[i].y) * 255) <<
               (unsigned char)(std::min(float(1), image[i].z) * 255);
    }
    ofs.close();
}

//[comment]
// In the main function, we will create the scene which is composed of some spheres
// and some light (which is also a sphere). Then, once the scene description is complete
// we render that scene, by calling the render() function.
//[/comment]
int main(int argc, char **argv)
{
    if (argc < 3) {
        std::cout << "No file detected" << '\n';
        return 1;
    }
    FILE* scene = NULL;
    scene = fopen(argv[1], "r");

    if (scene == NULL) {
        std::cout << "Error when reading file" << '\n';
    }
    unsigned width, height;
    unsigned s, l;
    fscanf(scene, "%u %u\n", &width, &height);
    fscanf(scene, "%u %u\n", &s, &l);
    Sphere* spheres = new Sphere[s+l];
    // spheres
    for(int i=0; i<s; i++) {
      float x, y, z, r, refl;
      fscanf(scene, "%f %f %f %f ", &x, &y, &z, &r);
      Vec3f position(x, y, z);
      fscanf(scene, "%f %f %f %f\n", &x, &y, &z, &refl);
      RGB color(x, y, z);
      // position, radius, surface color, reflectivity, transparency, emission color
      spheres[i] = Sphere(position, r, color, refl);
    }
    // lights
    for(int i=s; i<s+l; i++) {
      float x, y, z, r, refl, ec;
      fscanf(scene, "%f %f %f %f ", &x, &y, &z, &r);
      Vec3f position(x, y, z);
      fscanf(scene, "%f %f %f %f %f\n", &x, &y, &z, &refl, &ec);
      RGB color(x, y, z);
      // position, radius, surface color, reflectivity, transparency, emission color
      spheres[i] = Sphere(position, r, color, refl, ec);
    }
    Vec3f *image = new Vec3f[width * height];

    double runTime = rtclock();
    render(image, width, height, spheres, s+l);
    runTime = rtclock() - runTime;
    std::cout << "Run time: " << runTime << '\n';

    save(argv[2], image, width, height);

	delete image;
    delete spheres;

    return 0;
}
